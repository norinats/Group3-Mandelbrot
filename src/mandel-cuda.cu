#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

/**
 * writeOutput
 * 
 * Write Mandelbrot image in PGM format
 * 
 * @param fileName - Filename to write PGM data
 * @param data - output array data (Mandelbrot pixels)
 * @param width - image width
 * @param height - image height
 * 
 * */
 void writeOutput(const char *fileName, BYTE *data, int width, int height) {
    int i,j;
    int max = -1;
    int size = width*height;

    for (i=0; i < size; ++i) {
        if(data[i] > max) {
            max = data[i];
        }
    }

    FILE *fout = fopen(fileName, "w");

    fprintf(fout, "P2\n");
    fprintf(fout, "%d\t%d\n", width, height);
    fprintf(fout, "%d\n", max);

    for (i=0; i < height; ++i) {
        for (j=0; j<width; ++j) {
            fprintf(fout, "%d\t", data[i*width+j]);
        }
        fprintf(fout,"\n");
    }

    fflush(fout);
    fclose(fout);
}

void main() {
    
}